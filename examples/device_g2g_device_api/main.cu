#include "hip/hip_runtime.h"
/*
 * Minimal GPU-to-GPU example using UCX device-side API from a CUDA kernel.
 * Host sets up MPI + UCP, exchanges remote addresses and rkeys, creates a
 * ucp_device_mem_list, and launches a kernel that performs a device-side PUT.
 */

#include <mpi.h>
#include <hip/hip_runtime.h>
// #include <sm_60_atomic_functions.h>

#include <ucp/api/ucp.h>
#include <ucp/api/ucp_def.h>
#include <ucp/api/device/ucp_host.h>
#include <ucp/api/device/ucp_device_impl.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>
#include <stdexcept>

// Simple CUDA check
#define CUDA_CHECK(cmd) do { \
    hipError_t _e = (cmd); \
    if (_e != hipSuccess) { \
        fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(_e)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define MPI_CHECK(cmd) do { \
    int _e = (cmd); \
    if (_e != MPI_SUCCESS) { \
        fprintf(stderr, "MPI error %s:%d code=%d\n", __FILE__, __LINE__, _e); \
        MPI_Abort(MPI_COMM_WORLD, _e); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define UCP_CHECK(sts, msg) do { \
    if ((sts) != UCS_OK) { \
        fprintf(stderr, "UCX error %s:%d: %s failed: %d\n", __FILE__, __LINE__, (msg), (sts)); \
        MPI_Abort(MPI_COMM_WORLD, (sts)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// Kernel params
typedef struct {
    unsigned                              num_threads;
    unsigned                              num_blocks;
    ucs_device_level_t                    level;
    bool                                  with_request;
    const ucp_device_mem_list_handle_h   *mem_lists;  // device pointer to array of handles
    unsigned                              num_lists;  // equals world_size
} kernel_params_t;

// A single PUT operation descriptor for device-side API
typedef struct {
    unsigned     list_handle_index; // index into params.mem_lists (per-destination)
    unsigned     element_index;     // element inside that mem list (always 0 here)
    const void  *address;           // local source address on device
    uint64_t     remote_address;    // remote device address at peer
    size_t       length;            // bytes to transfer
} put_op_t;

template <ucs_device_level_t level>
__global__ void do_alltoallv_kernel(kernel_params_t params,
                                     const put_op_t *ops,
                                     unsigned num_ops,
                                     ucs_status_t *status_out)
{
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_ops) return;

    ucp_device_request_t req_obj;
    ucp_device_request_t *req = &req_obj;

    const put_op_t &op = ops[tid];
    ucs_status_t st = ucp_device_put_single<level>(params.mem_lists[op.list_handle_index],
                                                   op.element_index,
                                                   op.address,
                                                   op.remote_address,
                                                   op.length,
                                                   UCT_DEVICE_FLAG_NODELAY,
                                                   req);
    if (st != UCS_OK) {
        (void)atomicCAS((int*)status_out, (int)UCS_OK, (int)st);
    }
}

static void init_ucp(ucp_context_h &ucp_ctx, ucp_worker_h &worker)
{
    ucp_params_t ucp_params;
    memset(&ucp_params, 0, sizeof(ucp_params));
    ucp_params.field_mask = UCP_PARAM_FIELD_FEATURES;
    ucp_params.features   = UCP_FEATURE_RMA | UCP_FEATURE_DEVICE;
    UCP_CHECK(ucp_init(&ucp_params, nullptr, &ucp_ctx), "ucp_init");

    ucp_worker_params_t worker_params;
    memset(&worker_params, 0, sizeof(worker_params));
    worker_params.field_mask  = UCP_WORKER_PARAM_FIELD_THREAD_MODE;
    worker_params.thread_mode = UCS_THREAD_MODE_SINGLE;
    UCP_CHECK(ucp_worker_create(ucp_ctx, &worker_params, &worker), "ucp_worker_create");
}

static void create_all_endpoints(ucp_worker_h worker, int rank, int size, std::vector<ucp_ep_h> &eps)
{
    // Gather all worker addresses
    ucp_address_t *my_addr; size_t my_addr_len;
    UCP_CHECK(ucp_worker_get_address(worker, &my_addr, &my_addr_len), "ucp_worker_get_address");

    std::vector<size_t> addr_lens(size);
    MPI_CHECK(MPI_Allgather(&my_addr_len, 1, MPI_UNSIGNED_LONG, addr_lens.data(), 1, MPI_UNSIGNED_LONG, MPI_COMM_WORLD));

    // Convert size_t to int for MPI_Allgatherv
    std::vector<int> addr_lens_i(addr_lens.begin(), addr_lens.end());

    size_t total = 0; for (auto l : addr_lens) total += l;
    std::vector<uint8_t> all_addr(total);

    std::vector<int> displs(size, 0);
    for (int i = 1; i < size; ++i) displs[i] = displs[i-1] + addr_lens_i[i-1];

    MPI_CHECK(MPI_Allgatherv(my_addr, my_addr_len, MPI_BYTE,
                             all_addr.data(), addr_lens_i.data(), displs.data(), MPI_BYTE, MPI_COMM_WORLD));

    eps.assign(size, nullptr);
    for (int peer = 0; peer < size; ++peer) {
        if (peer == rank) continue;
        const uint8_t *peer_addr_ptr = all_addr.data() + displs[peer];
        ucp_ep_params_t ep_params; memset(&ep_params, 0, sizeof(ep_params));
        ep_params.field_mask = UCP_EP_PARAM_FIELD_REMOTE_ADDRESS;
        ep_params.address    = (const ucp_address_t*)peer_addr_ptr;
        UCP_CHECK(ucp_ep_create(worker, &ep_params, &eps[peer]), "ucp_ep_create");
    }

    ucp_worker_release_address(worker, my_addr);
}

int main(int argc, char **argv)
{
    // MPI Init
    MPI_CHECK(MPI_Init(&argc, &argv));
    int rank = 0, world_size = 0;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    if (world_size < 2) {
        if (rank == 0) fprintf(stderr, "Run with at least 2 ranks.\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }

    // Get local rank
    int local_rank = 0, local_size = 0;
    MPI_Comm local_comm;
    MPI_CHECK(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm));
    MPI_CHECK(MPI_Comm_rank(local_comm, &local_rank));
    MPI_CHECK(MPI_Comm_size(local_comm, &local_size));
    MPI_CHECK(MPI_Comm_free(&local_comm));

    // CUDA Init
    int ndev = 0;
    CUDA_CHECK(hipGetDeviceCount(&ndev));
    if (ndev == 0) {
        fprintf(stderr, "No CUDA devices available on rank %d\n", rank);
        MPI_Abort(MPI_COMM_WORLD, 1);
    }
    if (ndev < local_size) {
        fprintf(stderr, "Not enough CUDA devices available on rank %d\n", rank);
        MPI_Abort(MPI_COMM_WORLD, 1);
    }
    int dev = local_rank;
    CUDA_CHECK(hipSetDevice(dev));

    // Init UCP
    ucp_context_h ucp_ctx = nullptr; ucp_worker_h worker = nullptr;
    init_ucp(ucp_ctx, worker);

    // Create endpoints to all peers
    std::vector<ucp_ep_h> eps; eps.reserve(world_size);
    create_all_endpoints(worker, rank, world_size, eps);

    // Set sendcounts/displs (with variable sizes)
    size_t base_len = 1 << 20;
    std::vector<int> sendcounts(world_size, 0), senddispls(world_size, 0);
    for (int dst = 0; dst < world_size; ++dst) {
        size_t len = base_len + (((rank * world_size + dst) % 4) * 256);
        assert(len <= INT_MAX);
        sendcounts[dst] = len;
    }
    for (int i = 1; i < world_size; ++i) senddispls[i] = senddispls[i-1] + sendcounts[i-1];
    size_t total_send = senddispls.back() + sendcounts.back();

    // Compute recvcounts via alltoall
    std::vector<int> recvcounts(world_size, 0), recvdispls(world_size, 0);
    MPI_CHECK(MPI_Alltoall(sendcounts.data(), 1, MPI_INT, recvcounts.data(), 1, MPI_INT, MPI_COMM_WORLD));
    for (int i = 1; i < world_size; ++i) recvdispls[i] = recvdispls[i-1] + recvcounts[i-1];
    size_t total_recv = recvdispls.back() + recvcounts.back();

    // Allocate CUDA send/recv buffers
    void *send_buf = nullptr; void *recv_buf = nullptr;
    CUDA_CHECK(hipMalloc(&send_buf, total_send));
    CUDA_CHECK(hipMalloc(&recv_buf, total_recv));

    // Fill send segments with a dst-varying byte; clear recv
    for (int dst = 0; dst < world_size; ++dst) {
        void *seg = (void*)((uintptr_t)send_buf + senddispls[dst]);
        unsigned char dst_byte = (unsigned char)(0x10 + ((rank + dst) & 0xEF));
        CUDA_CHECK(hipMemset(seg, dst_byte, sendcounts[dst]));
    }
    CUDA_CHECK(hipMemset(recv_buf, 0x00, total_recv));

    // Register recv memory and pack rkey
    ucp_mem_map_params_t mmap_params; memset(&mmap_params, 0, sizeof(mmap_params));
    mmap_params.field_mask = UCP_MEM_MAP_PARAM_FIELD_ADDRESS | UCP_MEM_MAP_PARAM_FIELD_LENGTH | UCP_MEM_MAP_PARAM_FIELD_MEMORY_TYPE;
    mmap_params.address    = recv_buf;
    mmap_params.length     = total_recv;
    mmap_params.memory_type= UCS_MEMORY_TYPE_CUDA;

    ucp_mem_h recv_memh = nullptr;
    UCP_CHECK(ucp_mem_map(ucp_ctx, &mmap_params, &recv_memh), "ucp_mem_map(recv)");

    void *rkey_buf = nullptr; size_t rkey_size = 0;
    UCP_CHECK(ucp_rkey_pack(ucp_ctx, recv_memh, &rkey_buf, &rkey_size), "ucp_rkey_pack");

    // Share base remote address and rkey to all peers (variable sizes)
    uint64_t my_remote_addr = (uint64_t)recv_buf;
    struct { uint64_t addr; uint32_t size; } header { my_remote_addr, (uint32_t)rkey_size };

    std::vector<uint8_t> my_blob(sizeof(header) + rkey_size);
    memcpy(my_blob.data(), &header, sizeof(header));
    memcpy(my_blob.data() + sizeof(header), rkey_buf, rkey_size);

    std::vector<int> blob_sizes(world_size, 0), blob_displs(world_size, 0);
    MPI_CHECK(MPI_Allgather(&(header.size), 1, MPI_INT, blob_sizes.data(), 1, MPI_INT, MPI_COMM_WORLD));
    for (int i = 0; i < world_size; ++i) blob_sizes[i] += sizeof(header);
    for (int i = 1; i < world_size; ++i) blob_displs[i] = blob_displs[i-1] + blob_sizes[i-1];
    int total_blob = 0; for (int s : blob_sizes) total_blob += s;
    std::vector<uint8_t> all_blob(total_blob);
    MPI_CHECK(MPI_Allgatherv(my_blob.data(), my_blob.size(), MPI_BYTE,
                             all_blob.data(), blob_sizes.data(), blob_displs.data(), MPI_BYTE, MPI_COMM_WORLD));

    // Unpack each peer's rkey and record remote base address
    std::vector<ucp_rkey_h> peer_rkeys(world_size, nullptr);
    std::vector<uint64_t>   peer_bases(world_size, 0);
    for (int p = 0; p < world_size; ++p) {
        const uint8_t *ptr = all_blob.data() + blob_displs[p];
        struct { uint64_t addr; uint32_t size; } ph;
        memcpy(&ph, ptr, sizeof(ph));
        peer_bases[p] = ph.addr;
        const void *prkey = ptr + sizeof(ph);
        if (p != rank) {
            UCP_CHECK(ucp_ep_rkey_unpack(eps[p], prkey, &peer_rkeys[p]), "ucp_ep_rkey_unpack");
        }
    }

    // Gather each rank's recvdispls so senders can compute remote offsets
    std::vector<int> all_recvdispls(world_size * world_size, 0);
    MPI_CHECK(MPI_Allgather(recvdispls.data(), world_size, MPI_INT,
                            all_recvdispls.data(), world_size, MPI_INT, MPI_COMM_WORLD));

    // Map local send buffer for device-side API
    ucp_mem_h send_memh = nullptr;
    ucp_mem_map_params_t mmap_send; memset(&mmap_send, 0, sizeof(mmap_send));
    mmap_send.field_mask = UCP_MEM_MAP_PARAM_FIELD_ADDRESS | UCP_MEM_MAP_PARAM_FIELD_LENGTH | UCP_MEM_MAP_PARAM_FIELD_MEMORY_TYPE;
    mmap_send.address    = send_buf;
    mmap_send.length     = total_send;
    mmap_send.memory_type= UCS_MEMORY_TYPE_CUDA;
    UCP_CHECK(ucp_mem_map(ucp_ctx, &mmap_send, &send_memh), "ucp_mem_map(send)");

    // Create one device mem list per peer (excluding self), each with a single element
    std::vector<ucp_device_mem_list_handle_h> mem_lists(world_size, nullptr);
    std::vector<unsigned> element_index(world_size, 0);
    for (int p = 0; p < world_size; ++p) {
        if (p == rank) continue;
        ucp_device_mem_list_elem_t elem;
        elem.field_mask = UCP_DEVICE_MEM_LIST_ELEM_FIELD_MEMH | UCP_DEVICE_MEM_LIST_ELEM_FIELD_RKEY;
        elem.memh       = send_memh;
        elem.rkey       = peer_rkeys[p];

        ucp_device_mem_list_params_t ml_params;
        memset(&ml_params, 0, sizeof(ml_params));
        ml_params.field_mask   = UCP_DEVICE_MEM_LIST_PARAMS_FIELD_ELEMENTS |
                                 UCP_DEVICE_MEM_LIST_PARAMS_FIELD_ELEMENT_SIZE |
                                 UCP_DEVICE_MEM_LIST_PARAMS_FIELD_NUM_ELEMENTS;
        ml_params.element_size = sizeof(elem);
        ml_params.num_elements = 1;
        ml_params.elements     = &elem;

        ucs_status_t st;
        do {
            st = ucp_device_mem_list_create(eps[p], &ml_params, &mem_lists[p]);
            if (st == UCS_ERR_NOT_CONNECTED) {
                ucp_worker_progress(worker);
            }
        } while (st == UCS_ERR_NOT_CONNECTED);
        UCP_CHECK(st, "ucp_device_mem_list_create(per-peer)");
        element_index[p] = 0; // single element
    }

    // Build PUT operations: for each peer, compute local and remote offsets
    std::vector<put_op_t> ops; ops.reserve(world_size);
    for (int p = 0; p < world_size; ++p) {
        if (p == rank) continue;
        put_op_t op;
        op.list_handle_index = p;
        op.element_index     = element_index[p];
        op.address        = (const void*)((uintptr_t)send_buf + senddispls[p]);
        size_t remote_off = all_recvdispls[p * world_size + rank];
        op.remote_address = peer_bases[p] + remote_off;
        op.length         = sendcounts[p];
        ops.push_back(op);
    }

    // Upload ops to device
    put_op_t *d_ops = nullptr;
    CUDA_CHECK(hipMalloc(&d_ops, ops.size() * sizeof(put_op_t)));
    CUDA_CHECK(hipMemcpy(d_ops, ops.data(), ops.size() * sizeof(put_op_t), hipMemcpyHostToDevice));

    // Prepare kernel params
    kernel_params_t kparams = {};
    const unsigned threads_per_block = 128;
    const unsigned num_ops = static_cast<unsigned>(ops.size());
    const unsigned num_blocks = (num_ops + threads_per_block - 1) / threads_per_block;
    kparams.num_threads  = threads_per_block;
    kparams.num_blocks   = num_blocks;
    kparams.level        = UCS_DEVICE_LEVEL_THREAD;
    kparams.with_request = false;
    // Upload mem list handle array to device
    ucp_device_mem_list_handle_h *d_mem_lists = nullptr;
    CUDA_CHECK(hipMalloc(&d_mem_lists, sizeof(ucp_device_mem_list_handle_h) * world_size));
    CUDA_CHECK(hipMemcpy(d_mem_lists, mem_lists.data(), sizeof(ucp_device_mem_list_handle_h) * world_size, hipMemcpyHostToDevice));
    kparams.mem_lists  = d_mem_lists;
    kparams.num_lists  = world_size;

    // Launch kernel
    ucs_status_t *d_status = nullptr; ucs_status_t h_status = UCS_OK;
    CUDA_CHECK(hipMalloc(&d_status, sizeof(*d_status)));
    CUDA_CHECK(hipMemcpy(d_status, &h_status, sizeof(h_status), hipMemcpyHostToDevice));

    do_alltoallv_kernel<UCS_DEVICE_LEVEL_THREAD><<<kparams.num_blocks, kparams.num_threads>>>(kparams, d_ops, num_ops, d_status);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(&h_status, d_status, sizeof(h_status), hipMemcpyDeviceToHost));
    if (h_status != UCS_OK) {
        fprintf(stderr, "Rank %d kernel failed: %d\n", rank, h_status);
        MPI_Abort(MPI_COMM_WORLD, h_status);
    }
    // TODO: Is call to flush necessary?
    UCP_CHECK(ucp_worker_flush(worker), "ucp_worker_flush");

    // Handle self-copy on host to emulate alltoallv semantics for self
    void *dst = (void*)((uintptr_t)recv_buf + recvdispls[rank]);
    const void *src = (const void*)((uintptr_t)send_buf + senddispls[rank]);
    CUDA_CHECK(hipMemcpy(dst, src, sendcounts[rank], hipMemcpyDeviceToDevice));

    // Validate
    MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    for (int s = 0; s < world_size; ++s) {
        size_t len = recvcounts[s];
        std::vector<uint8_t> host_check(len, 0);
        const void *seg = (const void*)((uintptr_t)recv_buf + recvdispls[s]);
        CUDA_CHECK(hipMemcpy(host_check.data(), seg, len, hipMemcpyDeviceToHost));
        unsigned char expected = (unsigned char)(0x10 + ((s + rank) & 0xEF));
        for (size_t i = 0; i < len; ++i) {
            if (host_check[i] != expected) {
                fprintf(stderr, "Rank %d validation failed for segment from %d (expected 0x%02x)\n", rank, s, expected);
                MPI_Abort(MPI_COMM_WORLD, 1);
            }
        }
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_ops));
    CUDA_CHECK(hipFree(d_mem_lists));
    CUDA_CHECK(hipFree(d_status));
    for (int p = 0; p < world_size; ++p) {
        if (p == rank) continue;
        if (mem_lists[p]) ucp_device_mem_list_release(mem_lists[p]);
    }
    for (int p = 0; p < world_size; ++p) {
        if (peer_rkeys[p]) ucp_rkey_destroy(peer_rkeys[p]);
        if (p != rank && eps[p]) ucp_ep_destroy(eps[p]);
    }
    ucp_rkey_buffer_release(rkey_buf);
    UCP_CHECK(ucp_mem_unmap(ucp_ctx, send_memh), "ucp_mem_unmap(send)");
    UCP_CHECK(ucp_mem_unmap(ucp_ctx, recv_memh), "ucp_mem_unmap(recv)");
    ucp_worker_destroy(worker);
    ucp_cleanup(ucp_ctx);

    CUDA_CHECK(hipFree(send_buf));
    CUDA_CHECK(hipFree(recv_buf));

    MPI_CHECK(MPI_Barrier(MPI_COMM_WORLD));
    MPI_Finalize();

    return 0;
}



