#include "hip/hip_runtime.h"
/**
 * Copyright (c) NVIDIA CORPORATION & AFFILIATES, 2025. ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#include <hip/hip_runtime.h>
#include <ucp/device/lib.h>

#include <cstdint>

static __global__ void memcmp_kernel(const void* a, const void* b,
                                     int* result, size_t size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = idx; i < size; i += blockDim.x * gridDim.x) {
        if (reinterpret_cast<const uint8_t*>(a)[i]
            != reinterpret_cast<const uint8_t*>(b)[i]) {
            *result = 1;
            break;
        }
    }
}

// Compare generic CUDA buffers without copying them
int test_ucp_cuda_memcmp(const void *a, const void *b, size_t size)
{
    int *h_result, *d_result;
    int result;

    if ((hipHostAlloc(&h_result, sizeof(h_result), hipHostMallocMapped)
         != hipSuccess) ||
        (hipHostGetDevicePointer(&d_result, h_result, 0)
         != hipSuccess)) {
        return -1;
    }

    *h_result = 0;
    memcmp_kernel<<<16, 64>>>(a, b, d_result, size);
    hipDeviceSynchronize();
    result = *h_result;

    hipFree(h_result);
    return result;
}
