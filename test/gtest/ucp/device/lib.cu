#include "hip/hip_runtime.h"
/**
 * Copyright (c) NVIDIA CORPORATION & AFFILIATES, 2025. ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#include <hip/hip_runtime.h>
#include <ucp/device/lib.h>

#include <cstdint>

static __global__ void memcmp_kernel(const void* a, const void* b,
                                     int* result, size_t size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = idx; i < size; i += blockDim.x * gridDim.x) {
        if (reinterpret_cast<const uint8_t*>(a)[i]
            != reinterpret_cast<const uint8_t*>(b)[i]) {
            result[0] = 1;
            break;
        }
    }
}

// Compare generic CUDA buffers without copying them
int test_ucp_cuda_memcmp(const void *a, const void *b, size_t size)
{
    int h_result = 0;
    int* d_result;

    if (hipMalloc(&d_result, sizeof(*d_result)) != hipSuccess) {
        return -1;
    }

    if (hipMemcpy(d_result, &h_result, sizeof(h_result),
                   hipMemcpyHostToDevice)
        != hipSuccess) {
        h_result = -1;
        goto out;
    }

    memcmp_kernel<<<16, 64>>>(a, b, d_result, size);
    hipDeviceSynchronize();
    if (hipMemcpy(&h_result, d_result, sizeof(h_result),
                   hipMemcpyDeviceToHost)
        != hipSuccess) {
        h_result = -1;
    }

out:
    hipFree(d_result);
    return h_result;
}
