#include "hip/hip_runtime.h"
/**
 * Copyright (c) NVIDIA CORPORATION & AFFILIATES, 2025. ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#include "test_kernels.h"

#include <ucp/api/device/ucp_device_impl.h>
#include <ucs/debug/log.h>
#include <common/hip/hip_runtime.h>


template<ucs_device_level_t level>
ucs_status_t UCS_F_DEVICE
ucp_test_kernel_do_operation(const test_ucp_device_kernel_params_t &params,
                             uint64_t flags, ucp_device_request_t *req_ptr)
{
    ucs_status_t status;

    switch (params.operation) {
    case TEST_UCP_DEVICE_KERNEL_PUT_SINGLE:
        status = ucp_device_put_single<level>(params.mem_list,
                                              params.single.mem_list_index, 0,
                                              0, params.single.length, 0, flags,
                                              req_ptr);
        break;
    case TEST_UCP_DEVICE_KERNEL_PUT_MULTI:
        status = ucp_device_put_multi<level>(params.mem_list,
                                             params.multi.counter_inc_value, 0,
                                             flags, req_ptr);
        break;
    case TEST_UCP_DEVICE_KERNEL_PUT_MULTI_PARTIAL:
        status = ucp_device_put_multi_partial<level>(
                params.mem_list, params.partial.mem_list_indices,
                params.partial.mem_list_count,
                (size_t*)params.partial.local_offsets,
                (size_t*)params.partial.remote_offsets, params.partial.lengths,
                params.partial.counter_index, params.partial.counter_inc_value,
                params.partial.counter_remote_offset, 0, flags, req_ptr);
        break;
    case TEST_UCP_DEVICE_KERNEL_COUNTER_INC:
        status = ucp_device_counter_inc<level>(
                params.mem_list, params.counter_inc.mem_list_index,
                params.counter_inc.inc_value, 0, 0, flags, req_ptr);
        break;
    case TEST_UCP_DEVICE_KERNEL_COUNTER_WRITE:
        ucp_device_counter_write(params.local_counter.address,
                                 params.local_counter.value);
        /* req_ptr is not used in this case */
        return UCS_OK;
    case TEST_UCP_DEVICE_KERNEL_COUNTER_READ:
        uint64_t value = ucp_device_counter_read(params.local_counter.address);
        if (value != params.local_counter.value) {
            ucs_device_error("counter value mismatch: expected %lu, got %lu",
                             params.local_counter.value, value);
            return UCS_ERR_IO_ERROR;
        }
        /* req_ptr is not used in this case */
        return UCS_OK;
    }

    if (UCS_STATUS_IS_ERR(status)) {
        return status;
    }

    if (!(flags & UCT_DEVICE_FLAG_NODELAY) || (req_ptr == nullptr)) {
        return UCS_OK;
    }

    do {
        status = ucp_device_progress_req<level>(req_ptr);
    } while (status == UCS_INPROGRESS);
    return status;
}

template<ucs_device_level_t level> class device_request {
public:
    static constexpr size_t MAX_THREADS = 256;

    __device__ device_request(ucp_device_request_t *shared_reqs) :
        m_ptr(&shared_reqs[threadIdx.x / threads_per_req()])
    {
    }

    __device__ static constexpr size_t num_shared_reqs()
    {
        return MAX_THREADS / threads_per_req();
    }

    __device__ ucp_device_request_t *ptr() const
    {
        return m_ptr;
    }

private:
    __device__ static constexpr size_t threads_per_req()
    {
        switch (level) {
        case UCS_DEVICE_LEVEL_THREAD:
            return 1;
        case UCS_DEVICE_LEVEL_WARP:
            return UCS_DEVICE_NUM_THREADS_IN_WARP;
        default:
            return MAX_THREADS;
        }
    }

    ucp_device_request_t *m_ptr;
};

template <typename Func>
class scope_guard {
public:
    __device__ scope_guard(Func& func) : m_func(func) {}
    __device__ ~scope_guard() { m_func(); }

private:
    Func& m_func;
};

UCS_F_DEVICE ucs_status_t
ucp_test_kernel_get_state(const test_ucp_device_kernel_params_t &params,
                          test_ucp_device_kernel_result_t &result)
{
    uct_device_ep_t *device_ep;
    const uct_device_mem_element_t *uct_elem;
    uct_device_completion_t *comp;
    ucs_status_t status = UCS_OK;

    if (nullptr == params.mem_list) {
        return UCS_OK;
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        status = ucp_device_prepare_send(params.mem_list, 0, nullptr, device_ep,
                                         uct_elem, comp);
        if ((status == UCS_OK) &&
            (device_ep->uct_tl_id == UCT_DEVICE_TL_RC_MLX5_GDA)) {
            uct_rc_gdaki_dev_ep_t *ep =
                        reinterpret_cast<uct_rc_gdaki_dev_ep_t*>(device_ep);
            result.producer_index     = ep->sq_wqe_pi - result.producer_index;
            result.ready_index        = ep->sq_ready_index - result.ready_index;
            result.avail_count        = ep->avail_count - result.avail_count;
        }
    }

    __syncthreads();
    return status;
}

template<ucs_device_level_t level>
static __global__ void
ucp_test_kernel(const test_ucp_device_kernel_params_t params,
                test_ucp_device_kernel_result_t *result_ptr)
{
    /* Execute fence on any return, to ensure result is visible to the host */
    scope_guard fence(__threadfence_system);
    ucs_status_t &status = result_ptr->status;

    if (blockDim.x > device_request<level>::MAX_THREADS) {
        ucs_device_error("blockDim.x > MAX_THREADS");
        status = UCS_ERR_INVALID_PARAM;
        return;
    }

    __shared__ ucp_device_request_t
            shared_reqs[device_request<level>::num_shared_reqs()];
    device_request<level> req(shared_reqs);

    status = ucp_test_kernel_get_state(params, *result_ptr);
    if (status != UCS_OK) {
        return;
    }

    ucp_device_request_t *req_ptr = params.with_request ? req.ptr() : nullptr;
    uint64_t flags                = params.with_no_delay ?
                                                UCT_DEVICE_FLAG_NODELAY : 0;

    for (size_t i = 0; i < params.num_iters - 1; i++) {
        status = ucp_test_kernel_do_operation<level>(params, flags, req_ptr);
        if (status != UCS_OK) {
            return;
        }
    }

    // Last iteration must use no-delay flag and request, to be able to wait
    // properly for completion. Alternatively, we could add a device flush
    // function to the API.
    status = ucp_test_kernel_do_operation<level>(params, UCT_DEVICE_FLAG_NODELAY,
                                                 req.ptr());
    if (status != UCS_OK) {
        return;
    }

    status = ucp_test_kernel_get_state(params, *result_ptr);
}

static ucs_status_t check_warp_size()
{
    hipDevice_t cuda_device;
    hipError_t result;
    int warp_size;

    result = hipCtxGetDevice(&cuda_device);
    if (result != hipSuccess) {
        ucs_error("hipCtxGetDevice failed: %d", result);
        return UCS_ERR_NO_DEVICE;
    }

    result = hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize,
                                  cuda_device);
    if (result != hipSuccess) {
        ucs_error("hipDeviceGetAttribute failed: %d", result);
        return UCS_ERR_IO_ERROR;
    }

    if (UCS_DEVICE_NUM_THREADS_IN_WARP != warp_size) {
        ucs_error("Warp size mismatch: expected %d, got %d",
                  UCS_DEVICE_NUM_THREADS_IN_WARP, warp_size);
        return UCS_ERR_INVALID_PARAM;
    }

    return UCS_OK;
}

/**
 * Basic single element put operation.
 */
test_ucp_device_kernel_result_t
launch_test_ucp_device_kernel(const test_ucp_device_kernel_params_t &params)
{
    ucs_status_t check_status;

    check_status = check_warp_size();
    if (check_status != UCS_OK) {
        return {check_status};
    }

    ucx_cuda::device_result_ptr<test_ucp_device_kernel_result_t> result;
    result->status         = UCS_ERR_NOT_IMPLEMENTED;
    result->producer_index = 0;
    result->ready_index    = 0;
    result->avail_count    = 0;

    switch (params.level) {
    case UCS_DEVICE_LEVEL_THREAD:
        ucp_test_kernel<UCS_DEVICE_LEVEL_THREAD>
                <<<params.num_blocks, params.num_threads>>>(
                        params, result.device_ptr());
        break;
    case UCS_DEVICE_LEVEL_WARP:
        ucp_test_kernel<UCS_DEVICE_LEVEL_WARP>
                <<<params.num_blocks, params.num_threads>>>(
                        params, result.device_ptr());
        break;
    default:
        return {UCS_ERR_INVALID_PARAM};
    }

    ucs_status_t sync_status = ucx_cuda::synchronize();
    if (sync_status != UCS_OK) {
        return {sync_status};
    }

    return *result;
}
