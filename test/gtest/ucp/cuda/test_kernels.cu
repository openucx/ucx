#include "hip/hip_runtime.h"
/**
 * Copyright (c) NVIDIA CORPORATION & AFFILIATES, 2025. ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#include <cstdint>
#include <hip/hip_runtime.h>

#include "test_kernels.h"

namespace cuda {
static __global__ void memcmp_kernel(const void* s1, const void* s2,
                                     int* result, size_t size)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = idx; i < size; i += blockDim.x * gridDim.x) {
        if (reinterpret_cast<const uint8_t*>(s1)[i]
            != reinterpret_cast<const uint8_t*>(s2)[i]) {
            *result = 1;
            break;
        }
    }
}

// Compare generic CUDA buffers without copying them
int memcmp(const void *s1, const void *s2, size_t size)
{
    int *h_result, *d_result;
    int result;

    if (hipHostAlloc(&h_result, sizeof(*h_result), hipHostMallocMapped)
        != hipSuccess) {
        return -1;
    }

    if (hipHostGetDevicePointer(&d_result, h_result, 0) != hipSuccess) {
        result = 1;
        goto out;
    }

    *h_result = 0;
    memcmp_kernel<<<16, 64>>>(s1, s2, d_result, size);
    hipDeviceSynchronize();
    result = *h_result;

out:
    hipHostFree(h_result);
    return result;
}
}
