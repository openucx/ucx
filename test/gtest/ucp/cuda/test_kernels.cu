#include "hip/hip_runtime.h"
/**
 * Copyright (c) NVIDIA CORPORATION & AFFILIATES, 2025. ALL RIGHTS RESERVED.
 *
 * See file LICENSE for terms.
 */

#include "test_kernels.h"

#include <ucp/api/device/ucp_device_impl.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <memory>


namespace cuda {

/**
 * Wrapper class for a host memory result variable, that can be mapped to device
 * memory and passed to a Cuda kernel.
 */
template<typename T> class device_result_ptr {
public:
    device_result_ptr() : m_ptr(allocate(), release)
    {
    }

    device_result_ptr(const T &value) : m_ptr(allocate(), release)
    {
        *m_ptr.get() = value;
    }

    T &operator*()
    {
        return *m_ptr.get();
    }

    T *device_ptr()
    {
        T *device_ptr;
        if (hipHostGetDevicePointer(&device_ptr, m_ptr.get(), 0) !=
            hipSuccess) {
            throw std::runtime_error("hipHostGetDevicePointer() failure");
        }
        return device_ptr;
    }

private:
    static T *allocate()
    {
        T *ptr = nullptr;
        if (hipHostAlloc(&ptr, sizeof(T), hipHostMallocMapped) !=
            hipSuccess) {
            throw std::bad_alloc();
        }
        return ptr;
    }

    static void release(T *ptr)
    {
        hipHostFree(ptr);
    }

    std::unique_ptr<T, decltype(&release)> m_ptr;
};

static void synchronize()
{
    if (hipDeviceSynchronize() != hipSuccess) {
        throw std::runtime_error("hipDeviceSynchronize() failure");
    }
}

static __global__ void memcmp_kernel(const void* s1, const void* s2,
                                     int* result, size_t size)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    *result = 0;
    for (size_t i = idx; i < size; i += blockDim.x * gridDim.x) {
        if (reinterpret_cast<const uint8_t*>(s1)[i]
            != reinterpret_cast<const uint8_t*>(s2)[i]) {
            *result = 1;
            break;
        }
    }
}

static __global__ void
ucp_put_single_kernel(ucp_device_mem_list_handle_h mem_list,
                      const void *address, uint64_t remote_address,
                      size_t length, ucs_status_t *status)
{
    ucp_device_request_t req;
    ucs_status_t req_status;

    ucp_device_request_init(&req);
    req_status = ucp_device_put_single(mem_list, 0, address, remote_address,
                                       length, 0, &req);
    if (req_status != UCS_OK) {
        *status = req_status;
        return;
    }

    do {
        req_status = ucp_device_progress_req(&req);
    } while (req_status == UCS_INPROGRESS);
    *status = req_status;
}


/**
 * @brief Compares two blocks of device memory.
 *
 * Compares @a size bytes of the memory areas pointed to by @a s1 and @a s2,
 * which must both point to device memory.
 *
 * @param s1   Pointer to the first block of device memory.
 * @param s2   Pointer to the second block of device memory.
 * @param size Number of bytes to compare.
 *
 * @return int Returns 0 only if the memory blocks are equal.
 */
int launch_memcmp(const void *s1, const void *s2, size_t size)
{
    device_result_ptr<int> result = 0;

    memcmp_kernel<<<16, 64>>>(s1, s2, result.device_ptr(), size);
    synchronize();

    return *result;
}

/**
 * Basic single element put operation.
 */
ucs_status_t launch_ucp_put_single(ucp_device_mem_list_handle_h mem_list,
                                   const void *address, uint64_t remote_address,
                                   size_t length)
{
    device_result_ptr<ucs_status_t> status = UCS_ERR_NOT_IMPLEMENTED;

    ucp_put_single_kernel<<<1, 1>>>(mem_list, address, remote_address, length,
                                    status.device_ptr());
    synchronize();

    return *status;
}

} // namespace cuda
